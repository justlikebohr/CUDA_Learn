#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void hello_cuda() {
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;  //具体某个线程id
	printf("blockDim = [ %d ] block id = [ %d ], thread id = [ %d ] hello cuda\n", blockDim, blockIdx.x, idx);
}

int main() {
	hello_cuda<<< 1, 1 >>>();
	hipDeviceSynchronize();
	return 0;
}
